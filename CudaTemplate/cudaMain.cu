﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>
inline hipError_t checkCuda(hipError_t result, const char *error_step)
{
    if (result != hipSuccess) {
        printf("CUDA Step: %s\nCUDA Runtime Error: %s\n", error_step, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

int main()
{
    return 0;
}